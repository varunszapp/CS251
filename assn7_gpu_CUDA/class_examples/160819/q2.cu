
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void calculate(int *mem, int num, int offs)
{
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i-1 >= (num/(2*offs)))
           return;
      if((2*offs*i + offs) >= num)
        return;
      mem[2*offs*i] = mem[2*offs*i]^mem[2*offs*i + offs];

}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i,num;
    int *ptr;
    int *gpu_mem;
    int blocks;
    unsigned int seed;
    if(argc == 3){
         num = atoi(argv[1]);
         seed = atoi(argv[2]);
    }
    srand(seed);

    ptr = (int *)malloc(num*sizeof(int));
    for(i=0; i<num; ++i){
       ptr[i] = random();
    }

    gettimeofday(&t_start, NULL);

    hipMalloc(&gpu_mem, num * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");

    gettimeofday(&start, NULL);

    blocks = num /1024;

    if(num % 1024)
           ++blocks;

    for(int x=1;x<num;x*=2)
    {
        calculate<<<blocks, 1024>>>(gpu_mem, num, x);
    }

    CUDA_ERROR_EXIT("kernel invocation");
    gettimeofday(&end, NULL);

    hipMemcpy(ptr, gpu_mem, num * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);

    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);

    printf("result=%d\n",ptr[0]);

    free(ptr);
}
