
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))

__global__ void calculate(char *mem, int num,int skip)
{      
      int *arr=(int *)mem;   
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if((i*2*skip) >= num-1)
           return;
      *(arr+i*2*skip)=(*(arr+i*2*skip))^(*(arr+i*2*skip+skip));
      return;     
       
}

int main(int argc, char **argv)
{
    struct timeval start, end, t_start, t_end;
    int i,seed;
    char *ptr;
    char *sptr;
    int *pa;
    char *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int blocks;

    if(argc == 3){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
         seed=atoi(argv[2]);                  
    }

    srand(seed);

    /* Allocate host (CPU) memory and initialize*/

    ptr = (char *)malloc(num * sizeof(int));
    sptr = ptr; 
    for(i=0; i<num; ++i){
       pa = (int*) sptr;
       *pa=(int)rand();
       sptr +=sizeof(int);
    }
    
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

    hipMalloc(&gpu_mem, num * sizeof(int));
    CUDA_ERROR_EXIT("hipMalloc");

    hipMemcpy(gpu_mem, ptr, num * sizeof(int) , hipMemcpyHostToDevice);
    CUDA_ERROR_EXIT("hipMemcpy");
    
    gettimeofday(&start, NULL);
    
    blocks = num /1024;
    
    if(num % 1024)
           ++blocks;

    int skip=1;       
    for(i=num;i>1;){       
    calculate<<<blocks, 1024>>>(gpu_mem, num,skip);
    CUDA_ERROR_EXIT("kernel invocation");
    skip=skip*2;
    if(i%2==0)
        i=i/2;
    else i=i/2+1;    
    }
    gettimeofday(&end, NULL);
    
    /* Copy back result*/

    hipMemcpy(ptr, gpu_mem, num * sizeof(int) , hipMemcpyDeviceToHost);
    CUDA_ERROR_EXIT("memcpy");
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs Processsing =%ld microsecs\n", TDIFF(t_start, t_end), TDIFF(start, end));
    hipFree(gpu_mem);
    sptr = ptr;
   
    /*Print the answer*/ 
    pa = (int*)sptr;
    printf("result=%d\n", *pa);

    
    free(ptr);
}
